#include "hip/hip_runtime.h"
/**
 * This is a generic evaluator template. This file should be manually edited
 * and adapted to the problem at hand
 *
 * @author Mehran Maghoumi
 *
 */

/** =====================================Stack related definitions==================================== */
/** The size of the interpreter stack */
#define STACK_SIZE /*@@stack-size@@*/
#define push(A) do { sp++;stack[sp]=A; if(sp >= STACK_SIZE) printf("Stack overflow");} while(false)
#define pop(A) do{ A=stack[sp];sp--; if(sp < -1) printf("Stack underflow");} while(false)
/** ================================================================================================== */

/** The number of training instances that the individual is to be evaluated for */
#define PROBLEM_SIZE /*@@problem-size@@*/
#define BLOCK_SIZE /*@@block-size@@*/	// Used for the shared memory definitions


/************************************************************************************************************
 ************************************************************************************************************/

//TODO DOC: sadly there is only support for 1 pitch value for all input instances (which should be more than enough)
extern "C"
__global__ void evaluate(/*@@kernel-args@@*/ int inputPitch,
						/*@@kernel-out@@*/ int outputPitch,
						const char* __restrict__ individuals, const int indCounts, const int maxLength)
{
	int blockIndex = blockIdx.x;
	int threadIndex = threadIdx.x;

	if (blockIndex >= indCounts)
		return;

	// Obtain pointer to the beginning of the memory space of the individual that
	// this block will evaluate
	const char* __restrict__ expression = &(individuals[blockIndex * maxLength]);
	/*@@kernel-out-type@@*/ blockOutput = &(/*@@kernel-out-name@@*/[blockIndex * outputPitch]);

	// the first thread should reset these values
//	if (threadIndex == 0) {
//		fitnesses[blockIndex] = 0;
//	}

	/*@@kernel-out-type-nopointer@@*/ stack[STACK_SIZE];	// The stack is defined as the same type as the kernel output
	int sp;

	// Determine how many fitness cases this thread should process
	int portion = (PROBLEM_SIZE - 1)/ blockDim.x  + 1;

	for (int i = 0 ; i < portion; i++) {

		// Thread to data index mapping with respect to the loop variable
		int tid = portion * threadIndex + i;

		if (tid >= PROBLEM_SIZE)
			break;

		// Reset the stack pointer
		sp = - 1;

		int k = 0;	// Maintains the current index in the expression
		while(expression[k] != 0)
		{
			switch(expression[k])
			{
				/*@@interpreter@@*/
			}

			k++;
		}

		// Pop the top of the stack
		/*@@kernel-out-type-nopointer@@*/ stackTop;
		pop(stackTop);

		if(sp!=-1)
			printf("Stack pointer is not -1 but is %d", sp);

		// Assign the top of the stack to the output
		blockOutput[tid] = stackTop;

	}
}
